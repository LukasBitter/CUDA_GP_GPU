#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include <Device.h>
#include "Fractal.h"

using std::cout;
using std::endl;


/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
Fractal::Fractal(const Grid& grid, uint w, uint h, uint nMin, uint nMax, const DomaineMath& domaineMath) :
	Animable_I<uchar4>(grid, w, h, "Fractal_CUDA_rgba_uchar4",domaineMath), variateurAnimation(Interval<int>(nMin, nMax), 1)
    {
    // Tools
    this->t = nMin;					// protected dans super classe Animable

    }

Fractal::~Fractal(void)
    {
    // rien
    }

/*--------------------------------------*\
 |*		Surcharge			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void Fractal::process(uchar4* ptrTabPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("mandelbrot rgba uchar4 (before)"); // facultatif, for debug only, remove for release

    // TODO lancer le kernel avec <<<dg,db>>>
    // le kernel est importer ci-dessus (ligne 19)

    startFractal(ptrTabPixels,w,h, domaineMath);

    Device::lastCudaError("rippling rgba uchar4 (after)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 */
void Fractal::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
    }


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/



/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

