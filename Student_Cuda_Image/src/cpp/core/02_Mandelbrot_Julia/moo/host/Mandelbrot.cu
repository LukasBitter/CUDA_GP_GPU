#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include <Device.h>
#include "Mandelbrot.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

extern __global__ void mandelbrot(uchar4* ptrTabPixels,uint w, uint h, uint t, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
Mandelbrot::Mandelbrot(const Grid& grid, uint w, uint h, uint nMin, uint nMax, const DomaineMath& domaineMath) :
	Fractal(grid, w, h, nMin, nMax, domaineMath)
    {
    // Tools
    this->t = nMin;					// protected dans super classe Animable

    }

Mandelbrot::~Mandelbrot(void)
    {
    // rien
    }

/*--------------------------------------*\
 |*		Surcharge		*|
 \*-------------------------------------*/

void Mandelbrot::startFractal(uchar4* ptrTabPixels, uint w, uint h, const DomaineMath& domaineMath)
    {

    mandelbrot<<<dg,db>>>(ptrTabPixels,w,h,t, domaineMath);
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

