#include "hip/hip_runtime.h"
#include "Ray.h"

#include <iostream>
#include <assert.h>

#include <Device.h>

using std::cout;
using std::endl;


/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

extern __global__ void ray(uchar4* ptrTabPixels,uint w, uint h, uint t, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
Ray::Ray(const Grid& grid, uint w, uint h, uint nMin, uint nMax, const DomaineMath& domaineMath) :
	Animable_I<uchar4>(grid, w, h, "Ray_CUDA_rgba_uchar4",domaineMath), variateurAnimation(Interval<int>(nMin, nMax), 1)
    {
    // Tools
    this->t = nMin;					// protected dans super classe Animable

    }

Ray::~Ray(void)
    {
    // rien
    }

/*--------------------------------------*\
 |*		Surcharge			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void Ray::process(uchar4* ptrTabPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("ray rgba uchar4 (before)"); // facultatif, for debug only, remove for release

    // TODO lancer le kernel avec <<<dg,db>>>
    // le kernel est importer ci-dessus (ligne 19)

    ray<<<dg,db>>>(ptrTabPixels,w,h,t, domaineMath);

    Device::lastCudaError("rippling rgba uchar4 (after)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 */
void Ray::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
    }


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/



/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

