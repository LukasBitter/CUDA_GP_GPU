#include "hip/hip_runtime.h"
#include <iostream>

#include "Device.h"
#include "MonteCarlo.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void monteCarlo(float M, int nbFlechettes);
extern __global__ void setup_kernel_rand(hiprandState* tabDevGenerator,Device::getDeviceId() )

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

MonteCarlo::MonteCarlo(float M, int nbFlechettes) :
	M(M), nbFlechettes(nbFlechettes)
    {

    // Grid
	{
	this->dg = dim3(16, 2, 1); // disons, a optimiser selon le gpu
	this->db = dim3(32, 4, 1); // puissance de 2

	Device::gridHeuristic(dg, db);
	}

    // nb de blocks * nb de threads
    this->sizeOctetTabGenerator = dg.x * dg.y * db.z * db.x * db.y * db.z * sizeof(hiprandState);
    this->sizeOctetN0 = sizeof(int); // octet
    this->sizeSM = db.x * db.y * db.z * sizeof(int);

    // MM
	{
	// MM (malloc Device)
	    {
	    HANDLE_ERROR(hipMalloc(&ptrDevN0, sizeOctetN0));
	    HANDLE_ERROR(hipMalloc(&ptrTabDevGeneratorGM, sizeOctetTabGenerator));
	    }

	// MM (memset Device)
	    {
	    HANDLE_ERROR(hipMemset(ptrTabDevGeneratorGM, 0, sizeOctetN0));
	    }

	// MM (copy Host->Device)
	    {
	    // rien
	    }

	Device::lastCudaError("MonteCarlo MM (end allocation)"); // temp debug
	}
	setup_kernel_rand(tabDevGenerator,Device::getDeviceId());

    }

MonteCarlo::~MonteCarlo(void)
    {
    //MM (device free)
	{
	HANDLE_ERROR(hipFree(ptrDevN0));
	HANDLE_ERROR(hipFree(ptrTabDevGeneratorGM));

	Device::lastCudaError("MonteCarlo MM (end deallocation)"); // temp debug
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

public float getPi()
    {
    return this->pi;
    }

void MonteCarlo::run()
    {

    Device::lastCudaError("monteCarlo (before)"); // temp debug
    monteCarlo<<<dg,db, sizeSM>>>(ptrTabDevGeneratorGM, ptrDevN0, nbFlechettes); // assynchrone
    Device::lastCudaError("monteCarlo (after)"); // temp debug

    Device::synchronize(); // Temp, only for printf in  GPU

    // MM (Device -> Host)
	{
	HANDLE_ERROR(hipMemcpy(&n0, ptrDevN0, sizeOctetN0, hipMemcpyDeviceToHost)); // barriere synchronisation implicite
	}
	pi = (N0 / nbFlechettes ) * M // TODO à verifier
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
